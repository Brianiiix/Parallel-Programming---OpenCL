#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "helper.h"
extern "C"{
#include "hostFE.h"
}

__global__ void convKernel(int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage, float *outputImage){
    int halffilterSize = filterWidth / 2;
    float sum = 0;
    int k, l;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    for (k = -halffilterSize; k <= halffilterSize; k++){
	for (l = -halffilterSize; l <= halffilterSize; l++){
	    if (thisY + k >= 0 && thisY + k < imageHeight &&
		thisX + l >= 0 && thisX + l < imageWidth){
                sum += inputImage[(thisY + k) * imageWidth + thisX + l] *
                       filter[(k + halffilterSize) * filterWidth +
                              l + halffilterSize];
	    }
	}
    }

    int idx = thisX + thisY * imageWidth;
    outputImage[idx] = sum;
}

extern "C"
void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage, cl_device_id *device,
            cl_context *context, cl_program *program)
{
    int filterSize = filterWidth * filterWidth * sizeof(float);
    int ImgSize = imageHeight * imageWidth * sizeof(float);

    float *d_filter, *d_inputImage, *d_outputImage;
    hipMalloc(&d_filter, filterSize);
    hipMalloc(&d_inputImage, ImgSize);
    hipMalloc(&d_outputImage, ImgSize);

    hipMemcpy(d_filter, filter, filterSize, hipMemcpyHostToDevice);
    hipMemcpy(d_inputImage, inputImage, ImgSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlocks(imageWidth / threadsPerBlock.x, imageHeight / threadsPerBlock.y);
    convKernel<<<numBlocks, threadsPerBlock>>>(filterWidth, d_filter, imageHeight, imageWidth, d_inputImage, d_outputImage);

    hipMemcpy(outputImage, d_outputImage, ImgSize, hipMemcpyDeviceToHost);

    hipFree(d_filter);
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}
